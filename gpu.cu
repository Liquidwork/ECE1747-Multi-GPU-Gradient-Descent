#include "hip/hip_runtime.h"
//
// Created by Winter on 2023-11-20.
//

#include <iostream>
#include <vector>
#include <chrono>
#include "read.h"

#define THREADS_PER_BLOCK 512

using namespace std;

__global__ void compute(double *d_data, double *params, int *shape, double *d_mse, double *d_gradient) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    extern __shared__ double tmp[];
    double *line = &tmp[threadIdx.x * (shape[1] + 1)];

    double *row = &d_data[index * shape[1]];

    if (index >= shape[0]){
//        d_mse[blockIdx.x] = 0.;
//        for (int i = 0; i < shape[1]; i += 0){
//            d_gradient[blockIdx.x * shape[1] + i] = 0.;
//        }
        return;
    }

    double predict = 0.;

    for(int i = 0; i < shape[1] - 1; i++){
        predict += row[i] * params[i];
    }
    predict += params[shape[1] - 1];
    double error = predict - row[shape[1] - 1];
    line[shape[1]] = error * error / shape[0]; // mse
    for (int i = 0; i < shape[1] - 1; i++){
        line[i] = error * row[i] / shape[0]; // gradient
    }
    line[shape[1] - 1] = error / shape[0]; // gradient last term
    __syncthreads();


    // calculate partial sum
    for (unsigned int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (threadIdx.x >= stride && threadIdx.x < 2 * stride) {
            for (int i = 0; i < shape[1] + 1; i++){
                tmp[(threadIdx.x - stride) * (shape[1] + 1) + i] += line[i]; // sum grad and mse
            }
        }
        __syncthreads();
    }
    // Write the result of this block to global memory
    if (threadIdx.x == 0) {
        d_mse[blockIdx.x] = line[shape[1]];
        for (int i = 0; i < shape[1]; i++){
            d_gradient[blockIdx.x * shape[1] + i] = line[i];
        }
    }
}

int main(){
    const double learn_rate = 0.08;
    const int epoch = 1000;
    chrono::time_point<std::chrono::high_resolution_clock> start, stop;

    vector<double> data;
    int shape[2];
    readCSV("./data.csv", data, shape);

    cout << "Rows: " << shape[0] << endl;
    cout << "Columns: " << shape[1] << endl;
    double *d_data, *d_params;

    start = chrono::high_resolution_clock::now();

    double params[shape[1]] = {0};
    int *d_shape;
    hipMalloc((void**)&d_data, shape[0] * shape[1] * sizeof(double));
    hipMemcpy(d_data, data.data(), shape[0] * shape[1] * sizeof(double), hipMemcpyHostToDevice);

    hipMalloc((void**)&d_params, shape[1] * sizeof(double));

    hipMalloc((void**)&d_shape, 2 * sizeof(int));
    hipMemcpy(d_shape, shape, 2 * sizeof(int), hipMemcpyHostToDevice);

    int block = (shape[0] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;

    double *d_partial_mse, *d_partial_gradient;

    hipMalloc((void**)&d_partial_mse, block * sizeof(double));
    hipMalloc((void**)&d_partial_gradient, block * shape[1] * sizeof(double));

    double mse = 0;
    std::vector<double> gradient(shape[1], 0.);

    for(int i = 0; i < epoch; i++){
        hipMemcpy(d_params, params, shape[1] * sizeof(double), hipMemcpyHostToDevice);
        compute <<< block, THREADS_PER_BLOCK, THREADS_PER_BLOCK * (shape[1] + 1) * sizeof(double) >>>
                (d_data, d_params, d_shape, d_partial_mse, d_partial_gradient);

        double partial_mse[block], partial_gradient[block * shape[1]];

        hipMemcpy(partial_mse, d_partial_mse, block * sizeof(double), hipMemcpyDeviceToHost);
        hipMemcpy(partial_gradient, d_partial_gradient, block * shape[1] * sizeof(double), hipMemcpyDeviceToHost);

        mse = 0;
        for (int j = 0; j < block; j++){
            mse += partial_mse[j];
            for (int k = 0; k < shape[1]; k++){
                gradient[k] += partial_gradient[j * shape[1] + k];
            }

        }
        cout << "Epoch: " << i + 1 << "/" << epoch << ", Params: [";
        for (int k = 0; k < shape[1]; k++){
            cout << params[k] << ",";
            params[k] -= learn_rate * gradient[k];
            gradient[k] = 0.;
        }
        cout << "], MSE: " << mse << endl;
    }

    hipFree(d_data);
    hipFree(d_params);
    hipFree(d_shape);
    hipFree(d_partial_mse);
    hipFree(d_partial_gradient);

    stop = chrono::high_resolution_clock::now();
    double time = (double) chrono::duration_cast<chrono::nanoseconds>(stop - start).count() * 1e-9;
    cout << "Total time taken: " << time << "s." << endl;
}
